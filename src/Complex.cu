#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex>

#include "Complex.hpp"
#include "hipblas.h"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

class cublasHandle {
 public:
    cublasHandle() {
        hipblasStatus_t status = hipblasCreate(&this->_handle);
        assert(status == HIPBLAS_STATUS_SUCCESS);
    }

    ~cublasHandle() {
        hipblasDestroy(this->_handle);
    }

    operator hipblasHandle_t() { return this->_handle; }
 private:
    hipblasHandle_t _handle;
};

template<>
void Complex<gpuDouble>::_dmalloc() {
    this->_handle = new cublasHandle;
    hipMalloc(&this->_dptr, 2*this->_N*sizeof(Type));
    // helloCUDA<<<1, 1>>>();
    // hipDeviceSynchronize();
}

template<>
void Complex<gpuDouble>::_dfree() {
    if (this->_dptr) hipFree(this->_dptr);
    this->_dptr = nullptr;
    delete reinterpret_cast<cublasHandle*>(this->_handle);
}

template<>
void Complex<gpuDouble>::_memcpyHostToDevice() const {
    hipMemcpy(this->_dptr,
               this->_ptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyHostToDevice);
}

template<>
void Complex<gpuDouble>::_memcpyDeviceToHost() {
    hipMemcpy(this->_ptr,
               this->_dptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyDeviceToHost);
}

template<>
Complex<gpuDouble>& Complex<gpuDouble>::operator+=(const Complex<gpuDouble>& other) {
    this->_memcpyHostToDevice();
    other._memcpyHostToDevice();
    static const double alpha = 1.0;
    hipblasDaxpy(
        *reinterpret_cast<cublasHandle*>(this->_handle),  // handle
        2*this->_N,                                       // n
        &alpha,                                           // alpha
        reinterpret_cast<double*>(other._dptr),           // x
        1,                                                // incx
        reinterpret_cast<double*>(this->_dptr),            // y
        1);                                               // incy
    // hipDeviceSynchronize();
    this->_memcpyDeviceToHost();
    return *this;
}