#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex>

#include "Complex.hpp"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

template<>
Complex<gpuDouble>::Complex(size_t N) : _N(N) {
    this->_ptr = ::operator new(2*N*sizeof(Type));
    hipMalloc(&this->_dptr, 2*N*sizeof(Type));
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
}

template<>
Complex<gpuDouble>::Complex(std::initializer_list<double> l) : Complex(l.size()/2) {
    using Iter = typename std::initializer_list<Type>::const_iterator;
    Type* ptr = reinterpret_cast<Type*>(this->_ptr);
    for (Iter i = l.begin(); i < l.end(); i++) {
        *ptr++ = *i;
    }
    hipMemcpy(this->_dptr,
               this->_ptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyHostToDevice);
}

template<>
Complex<gpuDouble>::~Complex() {
    if (this->_ptr) ::operator delete(this->_ptr);
    if (this->_dptr) hipFree(this->_dptr);
}
