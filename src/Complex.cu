#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>

#include "Complex.hpp"
#include "hipblas.h"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

class CublasHandleSingleton {
 public:
    CublasHandleSingleton() {
        if (CublasHandleSingleton::_count++ == 0) {
            hipblasStatus_t status = hipblasCreate(&CublasHandleSingleton::_handle);
            assert(status == HIPBLAS_STATUS_SUCCESS);
        }
    }

    ~CublasHandleSingleton() {
        if (--CublasHandleSingleton::_count == 0) {
            hipblasDestroy(CublasHandleSingleton::_handle);
        }
    }

    operator hipblasHandle_t() { return this->_handle; }
 private:
    static hipblasHandle_t _handle;
    static int _count;
};

hipblasHandle_t CublasHandleSingleton::_handle;
int CublasHandleSingleton::_count = 0;

template<>
void Complex<gpuDouble>::_dmalloc() {
    this->_handle = new CublasHandleSingleton;
    hipMalloc(&this->_dptr, 2*this->_N*sizeof(Type));
    // helloCUDA<<<1, 1>>>();
    // hipDeviceSynchronize();
}

template<>
void Complex<gpuDouble>::_dfree() {
    if (this->_dptr) hipFree(this->_dptr);
    this->_dptr = nullptr;
    delete reinterpret_cast<CublasHandleSingleton*>(this->_handle);
}

template<>
void Complex<gpuDouble>::_memcpyHostToDevice() const {
    hipMemcpy(this->_dptr,
               this->_ptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyHostToDevice);
}

template<>
void Complex<gpuDouble>::_memcpyDeviceToHost() {
    hipMemcpy(this->_ptr,
               this->_dptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyDeviceToHost);
}

template<>
Complex<gpuDouble>& Complex<gpuDouble>::operator+=(const Complex<gpuDouble>& other) {
    this->_memcpyHostToDevice();
    other._memcpyHostToDevice();
    static constexpr double alpha = 1.0;
    hipblasDaxpy(
        *reinterpret_cast<CublasHandleSingleton*>(this->_handle),  // handle
        2*this->_N,                                                // n
        &alpha,                                                    // alpha
        reinterpret_cast<double*>(other._dptr),                    // x
        1,                                                         // incx
        reinterpret_cast<double*>(this->_dptr),                    // y
        1);                                                        // incy
    this->_memcpyDeviceToHost();
    return *this;
}

template<>
Complex<gpuDouble>& Complex<gpuDouble>::operator*=(const Complex<gpuDouble>& other) {
    this->_memcpyHostToDevice();
    other._memcpyHostToDevice();
    hipblasZdgmm(
        *reinterpret_cast<CublasHandleSingleton*>(this->_handle),  // handle
        HIPBLAS_SIDE_LEFT,                                          // mode
        this->_N,                                                  // m
        1,                                                         // n
        reinterpret_cast<hipDoubleComplex*>(other._dptr),           // A
        this->_N,                                                  // lda
        reinterpret_cast<hipDoubleComplex*>(this->_dptr),           // x
        1,                                                         // incx
        reinterpret_cast<hipDoubleComplex*>(this->_dptr),           // C
        this->_N);                                                 // ldc
    this->_memcpyDeviceToHost();
    return *this;
}
