#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex>

#include "Complex.hpp"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

template<>
Complex<gpuDouble>::Complex(size_t N) : _N(N) {
    this->_ptr = reinterpret_cast<void*>(::operator new(2*N*sizeof(Type)));
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
}

template<>
Complex<gpuDouble>::~Complex() {
    if (_ptr) ::operator delete(_ptr);
}
