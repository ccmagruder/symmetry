#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex>

#include "Complex.hpp"
#include "hipblas.h"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

template<>
void Complex<gpuDouble>::_dmalloc() {
    hipMalloc(&this->_dptr, 2*this->_N*sizeof(Type));
    // helloCUDA<<<1, 1>>>();
    // hipDeviceSynchronize();
}

template<>
void Complex<gpuDouble>::_dfree() {
    if (this->_dptr) hipFree(this->_dptr);
    this->_dptr = nullptr;
}

template<>
void Complex<gpuDouble>::_memcpyHostToDevice() const {
    hipMemcpy(this->_dptr,
               this->_ptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyHostToDevice);
}

template<>
void Complex<gpuDouble>::_memcpyDeviceToHost() {
    hipMemcpy(this->_ptr,
               this->_dptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyDeviceToHost);
}

template<>
Complex<gpuDouble>& Complex<gpuDouble>::operator+=(const Complex<gpuDouble>& other) {
    this->_memcpyHostToDevice();
    other._memcpyHostToDevice();
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double alpha(1);
    hipblasDaxpy(handle, 2*this->_N, &alpha, (double*)other._dptr, 1, (double*)this->_dptr, 1);
    hipblasDestroy(handle);
    this->_memcpyDeviceToHost();
    return *this;
}