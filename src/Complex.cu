#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex>

#include "Complex.hpp"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

template<>
Complex<gpuDoubleComplex>::Complex(size_t N) {
    this->_ptr = reinterpret_cast<void*>(new std::complex<double>[N]);
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
}

template<>
Complex<gpuDoubleComplex>::~Complex() {
    if (_ptr) delete [] reinterpret_cast<std::complex<double>*>(_ptr);
}
