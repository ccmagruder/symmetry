
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

int main()
{
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
