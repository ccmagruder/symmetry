#include "hip/hip_runtime.h"
// Copyright 2023 Caleb Magruder

#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex>

#include "Complex.hpp"

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

template<>
void Complex<gpuDouble>::_dmalloc() {
    hipMalloc(&this->_dptr, 2*this->_N*sizeof(Type));
    // helloCUDA<<<1, 1>>>();
    // hipDeviceSynchronize();
}

template<>
void Complex<gpuDouble>::_dfree() {
    if (this->_dptr) hipFree(this->_dptr);
    this->_dptr = nullptr;
}

template<>
void Complex<gpuDouble>::_memcpyHostToDevice() {
    hipMemcpy(this->_dptr,
               this->_ptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyHostToDevice);
}

template<>
void Complex<gpuDouble>::_memcpyDeviceToHost() {
    hipMemcpy(this->_ptr,
               this->_dptr,
               2 * this->_N * sizeof(Type),
               hipMemcpyDeviceToHost);
}
